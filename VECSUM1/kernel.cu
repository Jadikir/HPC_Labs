#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>
#include <random> 

float sum_cpu(const std::vector<float>& vec) {
    auto start = std::chrono::high_resolution_clock::now();
    float sum = std::accumulate(vec.begin(), vec.end(), 0.0f);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "CPU: Summ = " << sum << ", Time = " << duration.count() << " sec\n";
    return sum;
}

__global__ void sum_reduction(float* input, float* output, int n) {
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp_sum = (tid < n) ? input[tid] : 0.0f;
    cache[cacheIndex] = temp_sum;
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (cacheIndex < stride) {
            cache[cacheIndex] += cache[cacheIndex + stride];
        }
        __syncthreads();
    }
    if (cacheIndex == 0) {
        output[blockIdx.x] = cache[0];
    }
}

float sum_gpu_launcher(const std::vector<float>& vec) {
    int N = vec.size();

    float* d_input = nullptr;
    float* d_output = nullptr;
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::vector<float> partial_sums(numBlocks, 0.0f);
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, numBlocks * sizeof(float));
    hipMemcpy(d_input, vec.data(), N * sizeof(float), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    sum_reduction << <numBlocks, blockSize, blockSize * sizeof(float) >> > (d_input, d_output, N);
    hipDeviceSynchronize();
    if (numBlocks > 1) {
        int remainingBlocks = (numBlocks + blockSize - 1) / blockSize;
        sum_reduction << <remainingBlocks, blockSize, blockSize * sizeof(float) >> > (d_output, d_output, numBlocks);
        hipDeviceSynchronize();
        numBlocks = remainingBlocks;
    }
    float h_output = 0.0f;
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "GPU: Summ = " << h_output << ", Time = " << duration.count() << " sec\n";
    hipFree(d_input);
    hipFree(d_output);

    return h_output;
}
int main() {
    int N = 1000000000;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 10.0f);
    std::vector<float> vec(N);
    for (int i = 0; i < N; ++i) {
        vec[i] = dis(gen);
    }
    std::cout << N << "\n";
    float cpu_sum = sum_cpu(vec);
    float gpu_sum = sum_gpu_launcher(vec);
    float diff = std::abs(cpu_sum - gpu_sum);
    std::cout << "Difference between CPU and GPU results = " << diff << "\n";
    return 0;
}

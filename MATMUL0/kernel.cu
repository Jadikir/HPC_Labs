#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
const int N = 2000;

// CUDA Kernel для перемножения матриц
__global__ void matrixMulKernel(int* C, const int* A, const int* B, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int result = 0;
        for (int k = 0; k < width; ++k) {
            result += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = result;
    }
}

// Функция умножения матриц на GPU
void matrixMulCUDA(int* C, const int* A, const int* B, int width) {
    int* d_A, * d_B, * d_C;
    size_t size = width * width * sizeof(int);

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel <<<blocksPerGrid, threadsPerBlock >> > (d_C, d_A, d_B, width);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


void matrixMulCPU(int* C, const int* A, const int* B, int width) {
    for (int row = 0; row < width; ++row) {
        for (int col = 0; col < width; ++col) {
            C[row * width + col] = 0;
            for (int k = 0; k < width; ++k) {
                C[row * width + col] += A[row * width + k] * B[k * width + col];
            }
        }
    }
}


bool compareMatrices(const int* A, const int* B, int width) {
    for (int i = 0; i < width * width; ++i) {
        if (A[i] != B[i]) {
            return false;
        }
    }
    return true;
}

void measureTime(void(*func)(int*, const int*, const int*, int), int* C, const int* A, const int* B, int width, const char* description) {
    auto start = std::chrono::high_resolution_clock::now();
    func(C, A, B, width);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("%s took %f seconds\n", description, diff.count());
}

int main() {
    int* A = new int[N * N];
    int* B = new int[N * N];
    int* C = new int[N * N];
    int* C_CPU = new int[N * N];
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }
    measureTime(matrixMulCUDA, C, A, B, N, "CUDA");
    measureTime(matrixMulCPU, C_CPU, A, B, N, "CPU");
    if (compareMatrices(C, C_CPU, N)) {
        printf("Results are correct!\n");
    }
    else {
        printf("Results are incorrect!\n");
    }
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_CPU;
    return 0;
}